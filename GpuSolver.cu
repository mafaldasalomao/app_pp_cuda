#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include ""
#include "GpuSolver.h"


__global__ void AddKernel(int* c, const int* a, const int* b)
{
	int i = threadIdx.x;
	c[i] = a[i] + b[i];
}


using namespace std;
void useGPU::adding(int* c, const int* a, const int* b, unsigned int size) {
	int* dev_a = 0;
	int* dev_b = 0;
	int* dev_c = 0;
	hipMalloc((void**)&dev_c, size * sizeof(int));
	hipMalloc((void**)&dev_a, size * sizeof(int));
	hipMalloc((void**)&dev_b, size * sizeof(int));

	hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);

	hipDeviceSynchronize();

	AddKernel << <1, size >> > (dev_c, dev_a, dev_b);

	hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);

	hipFree(dev_c);
	hipFree(dev_a);
	hipFree(dev_b);
}